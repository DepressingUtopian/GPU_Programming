#pragma comment (lib,"cufft.lib")


#include "hip/hip_runtime.h"

#include <hipfft/hipfft.h>
#include <stdio.h>
#include <malloc.h>
#include <fstream>
#include <iostream>
#include <string>
#include <vector>
#include <algorithm>
#include <cctype>
#include <list>
#include <stdlib.h>


#define NX 64
#define BATCH 1
#define pi 3.141592

using namespace std;

int main()
{
	string line;
	hipfftHandle plan;
	hipfftComplex *cpu_data;
	hipfftComplex *gpu_data;
	std::vector<string> commands;
	vector<vector<string>> DATA;
	ifstream in("./data.txt");
	hipfftComplex *data_h;

	if (in.is_open())
	{
		while (getline(in, line))
		{
			
			std::string buffer = "";      //��������� ������
			for (int i = 0; i < line.size(); i++) {
				if (line[i] != ' ') {      // "�" ��������
					buffer += line[i];
				}
				else {
					if(buffer !="")
						commands.push_back(buffer);
					buffer = "";
				}
				if(i + 1 == line.size())
					commands.push_back(buffer);

			}
			if (commands.size() != 0)
			{
				DATA.push_back(commands);
				commands.clear();
			}
		}
	}
	in.close();

	hipMalloc((void**)&gpu_data, sizeof(hipfftComplex) *  DATA.size() * BATCH);
	data_h = (hipfftComplex*)calloc(DATA.size(), sizeof(hipfftComplex));
	cpu_data = new hipfftComplex[DATA.size() * BATCH];
	for (int i = 0; i < DATA.size() * BATCH; i++)
	{
		cpu_data[i].x = stof(DATA[i][2]);
		cpu_data[i].y = stof(DATA[i][3]);
	}
	hipMemcpy(gpu_data, cpu_data, sizeof(hipfftComplex) *  DATA.size() * BATCH, hipMemcpyHostToDevice);

	if (hipfftPlan1d(&plan, DATA.size() * BATCH, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS)
	{
		cerr << "ERROR hipfftPlan1d" << endl;
		return -1;
	}
	if (hipfftExecC2C(plan, gpu_data, gpu_data, HIPFFT_FORWARD) != HIPFFT_SUCCESS)
	{
		cerr << "ERROR hipfftPlan1d" << endl;
		return -1;
	}
	if (hipDeviceSynchronize() != HIPFFT_SUCCESS)
	{
		cerr << "ERROR hipfftPlan1d" << endl;
		return -1;
	}

	hipMemcpy(data_h,gpu_data, DATA.size() * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

	for (int i = 0; i < DATA.size(); i++)
		printf("%g\t%g\n",data_h[i].x,data_h[i].y);
   //�������� �������������� �������� ���!
	hipfftDestroy(plan);
	hipFree(gpu_data);
	free(data_h);
	free(cpu_data);

    return 0;
}
