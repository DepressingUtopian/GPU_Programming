#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <ctime>
#include <stdio.h>
#include <cstdlib>
#include <iostream>

#include <clocale>

float cuda_memory_malloc_test(int size, bool up);
float cuda_alloc_memory_malloc_test(int size, bool up);
//������ ��� ��������� ������
#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}
//���������
int countThreads = 1024; //��� �� �����
int countBlocks = 1000;

hipError_t addWithCuda(long  long int *c, const long  long int *a, const long  long int *b, long  long int size, unsigned int countBlock, unsigned int countThread);

static void RunVectorSumm(long  long int sizeVector, int countBlock, int countThread);

//������� ������������ �� GPU
__global__ void addKernel(long  long int *c, const  long  long int *a, const  long  long int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

const int SIZE = (10 * 1024 * 1024);

int main()
{
	int x;
	setlocale(LC_ALL, "Russian");
	srand(time(NULL));

	float elapsedTime;
	float MB = (float)100 * SIZE * sizeof(int)/1024/1024;
	
	elapsedTime = cuda_memory_malloc_test(SIZE, true);
	printf("����� ��� ������������� ����������� ������� ������ ��� ����������� �� GPU: %3.5f ms\n",elapsedTime);
	printf("\t��/� ��� ����������� �� GPU %3.1f\n", MB/(elapsedTime/1000));
	elapsedTime = cuda_memory_malloc_test(SIZE, false);
	printf("����� ��� ������������� ����������� ������� ������ ��� ����������� �� CPU: %3.5f ms\n", elapsedTime);
	printf("\t��/� ��� ����������� �� CPU %3.1f\n", MB / (elapsedTime / 1000));
	
	elapsedTime = cuda_alloc_memory_malloc_test(SIZE, true);
	printf("����� c �������������� ����������� ������� ������ ��� ����������� �� GPU: %3.5f ms\n", elapsedTime);
	printf("\t��/� ��� ����������� �� GPU %3.1f\n", MB / (elapsedTime / 1000));
	elapsedTime = cuda_alloc_memory_malloc_test(SIZE, false);
	printf("����� � �������������� ����������� ������� ������ ��� ����������� �� CPU: %3.5f ms\n", elapsedTime);
	printf("\t��/� ��� ����������� �� CPU %3.1f\n", MB / (elapsedTime / 1000));

		std::cout << "�����������: " << 100 << " ������: " << 10 << " �����: " << 10;
		countBlocks = (100 + countThreads - 1) / countThreads;
		RunVectorSumm(100, countBlocks, countThreads);



	return 0;
}

void RunVectorSumm(long  long int sizeVector, int countBlock, int countThread)
{
	//������������� �������
	hipStream_t stream0, stream1;
	long  long int arraySize = sizeVector;
	long  long int *a;
	long  long int *b;
	long  long int *c;


	hipHostAlloc((void**)&a, sizeVector * sizeof(*a), hipHostMallocDefault);
	hipHostAlloc((void**)&b, sizeVector * sizeof(*b), hipHostMallocDefault);
	hipHostAlloc((void**)&c, sizeVector * sizeof(*c), hipHostMallocDefault);

	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);

	for (int i = 0; i < arraySize; i++)
	{
		a[i] = rand();
		b[i] = rand();
	}

	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize, countBlock, countThread);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return;
	}
	//printf(" %d" + c[0]);

	/*printf("{");
	for (int i = 0; i < arraySize; i++)
	{
		printf(" %d" + c[i]);
	}
	printf("}");
	printf("\n");*/

	//������ ������� ���������� ��������� ��������� ����� ������� ��� ���������� ������ ������������ �������������� � ������������.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return;
	}

	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);
}
// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(long  long int *c, const long  long int *a, const long  long int *b, long  long int size, unsigned int countBlock, unsigned int countThread)
{
	long  long int *dev_a0 = 0;
	long  long int *dev_b0 = 0;
	long  long int *dev_c0 = 0;

	long  long int *dev_a1 = 0;
	long  long int *dev_b1 = 0;
	long  long int *dev_c1 = 0;

	hipStream_t stream0, stream1;
	hipError_t cudaStatus;
	hipEvent_t start, stop;


	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);
	
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);



	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c0, size * sizeof(long  long int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a0, size * sizeof(long  long int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b0, size * sizeof(long  long int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_c1, size * sizeof(long  long int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a1, size * sizeof(long  long int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b1, size * sizeof(long  long int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	hipEventSynchronize(start);
	hipEventRecord(start, 0);

	for (int i = 0; i < size - 1; i+=2)
	{
		//printf("\n %d",i);
		//printf("\n %d", i + 1);
		cudaStatus = hipMemcpy(dev_a0, a + i, 1 * sizeof(long  long int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		cudaStatus = hipMemcpy(dev_a1, a + i + 1, 1 * sizeof(long  long int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_b0, b + i, 1 * sizeof(long  long int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		cudaStatus = hipMemcpy(dev_b1, b + i + 1, 1 * sizeof(long  long int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		addKernel << <1,1, 1,stream0 >> > (dev_c0, dev_a0, dev_b0);
		addKernel << <1,1, 1, stream1 >> > (dev_c1, dev_a1, dev_b1);
		
		hipMemcpyAsync(c + i, dev_c0, 1 * sizeof(long long int), hipMemcpyDeviceToHost, stream0);
		hipMemcpyAsync(c + i + 1, dev_c1, 1 * sizeof(long long int), hipMemcpyDeviceToHost, stream1);
	}
	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);

	//������ �������� Event
	
	//��������� ���������� �� GPU countBlock - ���������� ������(������) , countThread - ���������� �����
	
	hipEventRecord(stop, 0);
	hipDeviceSynchronize();
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	std::cout << std::endl;
	printf("����� ����������: %f ms\n", time);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}


	cudaCheckError(hipDeviceSynchronize());

	cudaCheckError(cudaStatus = hipMemcpy(c, dev_c, size * sizeof(long  long int), hipMemcpyDeviceToHost));


	//��������� �������� Event


	//����� ���������� ����������
	/*
	printf("{");
	for (int i = 0; i < size; i++)
	{
		printf(" %d", a[i]);

	}
	printf("} + ");
	printf("{");
	for (int i = 0; i < size; i++)
	{

		printf(" %d", b[i]);

	}
	printf("} = ");
	printf("{");
	for (int k = 0; k < size; k++)
	{
		printf(" %d", c[k]);
	}
	printf("}");*/

Error:
	hipFree(dev_c0);
	hipFree(dev_a0);
	hipFree(dev_b0);
	hipFree(dev_c1);
	hipFree(dev_a1);
	hipFree(dev_b1);

	hipStreamDestroy(stream0);
	hipStreamDestroy(stream1);

	return cudaStatus;
}
float cuda_memory_malloc_test(int size,bool up)
{
	hipEvent_t start, stop;
	hipError_t cudaStatus;

	int *a, *dev_a;
	float elapsedTime = 0.0f;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	a = (int*)malloc(size * sizeof(*a));
	hipMalloc((void**)&dev_a, size * sizeof(*dev_a));
	

	hipEventSynchronize(start);
	hipEventRecord(start, 0);

	for (int i = 0; i < 100; i++) {
		if (up)
		{
			hipMemcpy(dev_a, a, size * sizeof(*dev_a), hipMemcpyHostToDevice);
		}
		else
			hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost);
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);


	free(a);

	hipFree(dev_a);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return elapsedTime;
}
float cuda_alloc_memory_malloc_test(int size, bool up)
{
	hipEvent_t start, stop;
	int *a, *dev_a;
	float elapsedTime = 0.0f;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc((void**)&dev_a, size * sizeof(*dev_a));
	hipHostAlloc((void**)&a, size * sizeof(*a),hipHostMallocDefault);

	hipEventSynchronize(start);
	hipEventRecord(start, 0);

	for (int i = 0; i < 100; i++) {
		if (up)
		{
			hipMemcpy(dev_a, a, size * sizeof(*a), hipMemcpyHostToDevice);
		}
		else
			hipMemcpy(a, dev_a, size * sizeof(*a), hipMemcpyDeviceToHost);
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipFree(dev_a);
	hipHostFree(a);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return elapsedTime;
}