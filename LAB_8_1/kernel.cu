
#pragma comment (lib,"cublas.lib")

#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>

#include <hipblas.h>
#include <hipblas.h>



using namespace std;

#define N 1<<10
#define V 0.2
#define T 2
struct functor {
	const float koef;
	functor(float _koef) : koef(_koef) {}
	__host__ __device__ float operator()(float x, float y) { return koef * x + y; }
};
void saxpy(float _koef, thrust::device_vector<float> &x,
	thrust::device_vector<float> &y)
{
	functor func(_koef);
	thrust::transform(x.begin(), x.end(), y.begin(), y.begin(), func);

}


int main()
{
	setlocale(LC_ALL, "Russian");
	float Function[N];
	float FunctionData[N];
	hipEvent_t start, stop;
	float *x = new float [N];
	float *y = new float [N];
	thrust::host_vector<float> cpumem1(N);
	thrust::host_vector<float> cpumem2(N);

	float *dev_x;
	float *dev_y;

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	float alpha = V * T;

	float time;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc(&dev_x, N);
	hipMalloc(&dev_y, N);

	for (int i = 0; i < N; i++) {
		FunctionData[i] = rand() % 100;
	}


	

	for (int i = 0; i < N; i++)
	{
		cpumem1[i] = FunctionData[i];

		(i - 1 >= 0) ? cpumem2[i] = FunctionData[i - 1] : cpumem2[i] = FunctionData[N - 1];
	}
	thrust::device_vector<float> gpumem1 = cpumem1;
	thrust::device_vector<float> gpumem2 = cpumem2;
	for (int i = 0; i < N; i++)
	{
		x[i] = cpumem1[i];
		y[i] = cpumem2[i];
	}
	hipEventSynchronize(start);
	

	hipEventRecord(start, 0);

	saxpy(V*T, gpumem1, gpumem1);

	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	std::cout << std::endl;

	printf("Thrust\n");
	printf("����� ����������: %f ms\n", time);
	//for (int i = 0; i < N; i++)
	//	cout << gpumem1[i] << " ";

	cublasInit();

	hipblasSetVector(N, sizeof(x[0]), x, 1, dev_x, 1);
	hipblasSetVector(N, sizeof(y[0]), y, 1, dev_y, 1);

	hipEventSynchronize(start);
	hipEventRecord(start, 0);

	hipblasSaxpy(handle, N, &alpha, dev_x, 1, dev_y, 1);

	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	hipblasGetVector(N, sizeof(y[0]), dev_y, 1, y, 1);
	cublasShutdown();

	std::cout << std::endl;
	printf("cuBLAS\n");
	printf("����� ����������: %f ms\n", time);
	
	free(x);
	free(y);
	hipFree(dev_x);
	hipFree(dev_y);
	hipblasDestroy(handle);
	return 0;
}

float TransportEquation()
{
	return 0;
}