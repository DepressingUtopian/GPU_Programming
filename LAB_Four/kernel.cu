#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <clocale>
#include <stdio.h>

#include <cuda_occupancy.h>


const int sizeShared = 32;
//const int arraySize = 30484848;
//������ ��� ��������� ������
#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}

__global__ void MatrixInicialization(float *_matrix)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	int N = blockDim.x * gridDim.x;

	_matrix[i + j * N] = (float)(i + j * N);
}

//NVIDIA DOCS 1
__global__ void coalescedMultiply(float *a, float* b, float *c,
	int N)
{
	__shared__ float aTile[sizeShared][sizeShared];

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	float sum = 0.0f;
	aTile[threadIdx.y][threadIdx.x] = a[row*sizeShared + threadIdx.x];
	for (int i = 0; i < sizeShared; i++) {
		sum += aTile[threadIdx.y][i] * b[i*N + col];
	}
	c[row*N + col] = sum;
}
//NVIDIA DOCS 1
__global__ void sharedABMultiply(float *a, float* b, float *c,
	int N)
{
	__shared__ float aTile[sizeShared][sizeShared],
		bTile[sizeShared][sizeShared];
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	float sum = 0.0f;
	aTile[threadIdx.y][threadIdx.x] = a[row*sizeShared + threadIdx.x];
	bTile[threadIdx.y][threadIdx.x] = b[threadIdx.y*N + col];
	__syncthreads();
	for (int i = 0; i < sizeShared; i++) {
		sum += aTile[threadIdx.y][i] * bTile[i][threadIdx.x];
	}
	c[row*N + col] = sum;
}
__global__ void MatrixTranspose(float *_matrix1, float *_matrix2)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	int N = blockDim.x * gridDim.x;

	
	_matrix1[j + i * N] = _matrix2[i + j * N];
}
__global__ void MatrixTranspose_with_SharedMemory(float *_matrix1, float *_matrix2)
{
	__shared__ float tempMemory[sizeShared][sizeShared];

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	int N = blockDim.x * gridDim.x;

	tempMemory[threadIdx.y][threadIdx.x] = _matrix2[i + j * N];
	//printf("%d \n", i + j * N);
	__syncthreads();

	i = threadIdx.x + blockIdx.x * blockDim.x;
	j = threadIdx.y + blockIdx.y * blockDim.y;
	
	_matrix1[i + j * N] = tempMemory[threadIdx.x][threadIdx.y];
}
__global__ void MatrixTranspose_with_SharedMemoryCoalising(float *_matrix1, float *_matrix2)
{
	__shared__ float tempMemory[sizeShared][sizeShared + 1]; //������� ��������

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	int N = blockDim.x * gridDim.x;

	tempMemory[threadIdx.y][threadIdx.x] = _matrix2[i + j * N];
	//printf("%d \n", i + j * N);
	__syncthreads();

	i = threadIdx.x + blockIdx.x * blockDim.x;
	j = threadIdx.y + blockIdx.y * blockDim.y;

	_matrix1[i + j * N] = tempMemory[threadIdx.x][threadIdx.y];
}
void ViewMatrix(float *a, int size);
void MatrixTest(int size, int countThread);

using namespace std;

int main()
{
	setlocale(LC_ALL, "Russian");
	
	MatrixTest(2048,32);
	cout << "33" << endl;
	return 0;
}

void MatrixTest(int size,int countThread)
{

	int gridSize = size / countThread;

	if (size % countThread)
	{
		cout << "������ ���������� ������� ������ ���� ������" << endl;
	}
	if (countThread > size)
	{
		cout << "������� ����� �������" << endl;
	}
	float *a = new float[size * size];
	float *b = new float[size * size];
	float *c = new float[size * size];


	float *dev_a = nullptr;
	float *dev_b = nullptr;
	float *dev_c = nullptr;

	hipError_t cudaStatus;
	for(int i = 0;i < size * size;i++)	
		{
			a[i] = 1;
			b[i] = 1;
		}

	cudaStatus = hipMalloc((void**)&dev_a, size * size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");

	}

	cudaStatus = hipMemcpy(dev_a, a, size * size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");

	}
	cudaStatus = hipMalloc((void**)&dev_b, size * size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");

	}

	cudaStatus = hipMemcpy(dev_b, b, size * size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");

	}


	cudaStatus = hipMalloc((void**)&dev_c, size * size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");

	}

	cudaStatus = hipMemcpy(dev_c, c, size * size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");

	}
	sharedABMultiply << <dim3(gridSize, gridSize), dim3(countThread, countThread) >> > (dev_a,dev_b,dev_c,size);
	hipDeviceSynchronize();
	/*MatrixInicialization << <dim3(gridSize, gridSize), dim3(countThread, countThread) >> > (dev_a);
	MatrixTranspose_with_SharedMemory << < dim3(gridSize, gridSize), dim3(countThread, countThread) >> > (dev_c, dev_a);
	hipDeviceSynchronize();
	MatrixInicialization << < dim3(gridSize, gridSize), dim3(countThread, countThread) >> > (dev_a);
	MatrixTranspose_with_SharedMemoryCoalising << < dim3(gridSize, gridSize), dim3(countThread, countThread) >> > (dev_c, dev_a);
	hipDeviceSynchronize();*/

	cudaStatus = hipMemcpy(a, dev_a, size * size * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");

	}
	cudaStatus = hipMemcpy(b, dev_b,size * size * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");

	}
	cudaStatus = hipMemcpy(c, dev_c, size * size * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");

	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	ViewMatrix(c, size);
}

void ViewMatrix(float *a, int size)
{
	cout << endl;
	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
			if ((j + i * size) % size == 0)
				cout << a[j + i * size] << " ";
			else
				cout << "\t" << a[j + i * size] << " ";

		cout << endl;
	}

	cout << endl;
}