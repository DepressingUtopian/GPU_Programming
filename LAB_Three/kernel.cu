
#include "hip/hip_runtime.h"


#include <iostream>
#include <clocale>
#include <stdio.h>

//������ ��� ��������� ������
#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}
//���������

const int countThreads = 1024; //��� �� �����
const int countBlocks = 1;

void getViewGPUSpec();

using namespace std;

int main()
{
	setlocale(LC_ALL,"Russian");
	getViewGPUSpec();
    return 0;
}

void getViewGPUSpec()
{
	int count = 0;
	int dev;
	hipDeviceProp_t prop;

	hipError_t error_id = hipGetDeviceCount(&count);//�������� ���������� ��������� GPU

	if (error_id != hipSuccess) {
		printf("hipGetDeviceCount returned %d\n-> %s\n",
			static_cast<int>(error_id), hipGetErrorString(error_id));
		printf("Result = FAIL\n");
		exit(EXIT_FAILURE);
	}

		

		for (dev = 0; dev < count; ++dev) {

			hipSetDevice(dev);
			hipGetDeviceProperties(&prop, dev);
			cout << "��������� ���������� �� ���������� : " << prop.name << endl;
			cout << "����� ���������� ������ : " << prop.totalGlobalMem / 1024 / 1024 << " MB" << endl;
			cout << "����� ����������� ������ � ����� ����� : " << prop.sharedMemPerBlock << " ���� " << endl;
			cout << "���������� ����� � �����  : " << prop.warpSize << endl;
			cout << "���������� �����������������  : " << prop.multiProcessorCount << endl;
			cout << "����� ����������� ������  : " << prop.totalConstMem << endl;
			cout << "������������ ���������� ����������������� �� ���� : " << prop.maxThreadsPerMultiProcessor << endl;
			cout << "������������ ���������� ������� �� ����  : " << prop.maxThreadsPerBlock << endl;

		}
}

