#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <clocale>
#include <stdio.h>

#include <cuda_occupancy.h>



//const int arraySize = 30484848;
//������ ��� ��������� ������
#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}

__global__ void MatrixInicialization(float *_matrix)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	int N = blockDim.x * gridDim.x;

	_matrix[i + j * N] = (float)(i + j * N);
}

__global__ void MatrixInicializationBlock(float *_matrix)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	int N = blockDim.x * gridDim.x;

	_matrix[i + j * N] = (float)(i + j * N);
}
					
__global__ void MatrixTranspose(float *_matrix1, float *_matrix2)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	int N = blockDim.x * gridDim.x;

	_matrix1[j + i * N] = _matrix2[i + j * N];
}

void getViewGPUSpec();
void Occupancy(int N);
void ViewMatrix(float *a, int size);
void MatrixTest(int size,void* func);

using namespace std;

int main()
{
	setlocale(LC_ALL,"Russian");
	getViewGPUSpec();

	for (long long int i = 10; i < 10e4; i <<= 1)
	{
		Occupancy(i);
	}
	MatrixTest(20, MatrixInicialization);

    return 0;
}

void getViewGPUSpec()
{
	int count = 0;
	int dev;
	hipDeviceProp_t prop;

	hipError_t error_id = hipGetDeviceCount(&count);//�������� ���������� ��������� GPU

	if (error_id != hipSuccess) {
		printf("hipGetDeviceCount returned %d\n-> %s\n",
			static_cast<int>(error_id), hipGetErrorString(error_id));
		printf("Result = FAIL\n");
		exit(EXIT_FAILURE);
	}

		

		for (dev = 0; dev < count; ++dev) {

			hipSetDevice(dev);
			hipGetDeviceProperties(&prop, dev);
			cout << "��������� ���������� �� ���������� : " << prop.name << endl;
			cout << "����� ���������� ������ : " << prop.totalGlobalMem / 1024 / 1024 << " MB" << endl;
			cout << "����� ����������� ������ � ����� ����� : " << prop.sharedMemPerBlock << " ���� " << endl;
			cout << "���������� ����� � �����  : " << prop.warpSize << endl;
			cout << "���������� �����������������  : " << prop.multiProcessorCount << endl;
			cout << "����� ����������� ������  : " << prop.totalConstMem << endl;
			cout << "������������ ���������� ����� ��  ��������������� : " << prop.maxThreadsPerMultiProcessor << endl;
			cout << "������������ ���������� ����� �� ����  : " << prop.maxThreadsPerBlock << endl;

			cout <<endl << "������������� ������������� � ����� ����� � �����" << endl;
			cout << "����������� ���������� ������ � ����� :" << prop.maxThreadsPerMultiProcessor / prop.warpSize << endl;
			cout << "����������� ���������� ����� � ����� :" << prop.maxThreadsPerMultiProcessor / prop.multiProcessorCount << endl;
		

		}
}

void Occupancy(int N)
{
	int blockSize = 0;
	int numBlocks = 0;
	int minGridSize = 0;
	int activeWarps = 0;
	int maxWarps = 0;

	int arraySize = N;
	int device;

	int gridSize;

	float *a = new float[arraySize];
	float *b = new float[arraySize];

	float *dev_a = 0;
	float *dev_b = 0;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);

	hipDeviceProp_t prop;

	hipGetDevice(&device);
	hipGetDeviceProperties(&prop, device);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");

	}
	cudaStatus = hipMalloc((void**)&dev_a, arraySize * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");

	}

	cudaStatus = hipMalloc((void**)&dev_b, arraySize * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");

	}

	cudaStatus = hipMemcpy(dev_a, a, arraySize * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");

	}

	cudaStatus = hipMemcpy(dev_b, b, arraySize * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");

	}


	hipOccupancyMaxPotentialBlockSize(
		&minGridSize,
		&blockSize,
		(void*)MatrixInicialization,
		0,
		arraySize);

	hipOccupancyMaxActiveBlocksPerMultiprocessor(
		&numBlocks,
		MatrixInicialization,
		blockSize,
		0);


	// Round up according to array size
	gridSize = (arraySize + blockSize - 1) / blockSize;



	//gTest2 <<<countBlocks, countThreads >>> (dev_b);
	//hipDeviceSynchronize();

	activeWarps = numBlocks * blockSize / prop.warpSize;
	maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize;

	cout << "������� ��� N -������� �������" << endl;
//	cout << "����������� ���������� ������  ��� N = " << arraySize << " , = " <<  (double)gridSize / ((gridSize + blockSize - 1) / gridSize) << endl;
	cout << "����������� ���������� ������   ��� N = " << arraySize << " , = " << gridSize << endl;
	cout << "����������� ���������� �����   ��� N = " << arraySize << " , = " << blockSize << endl;
	std::cout << "Occupancy: " << (double)activeWarps / maxWarps * 100 << "%" << std::endl;
	std::cout << "Occupancy: " << (double)((blockSize * prop.multiProcessorCount) / (gridSize * maxWarps)) * 100 << "%" << std::endl;
	MatrixInicialization <<<gridSize, blockSize >>> (dev_a);
	hipDeviceSynchronize();

Error:
	hipFree(dev_a);
	hipFree(dev_b);
}

void getOccupancy(int N,int &_gridSize,int &_blockSize, void* func)
{
	int blockSize = 0;
	int numBlocks = 0;
	int minGridSize = 0;
	int activeWarps = 0;
	int maxWarps = 0;

	int arraySize = N;
	int device;

	int gridSize;


	hipError_t cudaStatus;


	hipOccupancyMaxPotentialBlockSize(
		&minGridSize,
		&blockSize,
		MatrixInicialization,
		0,
		N);

	// Round up according to array size
	gridSize = (N + blockSize - 1) / blockSize;

	_gridSize = gridSize;
	_blockSize = blockSize;
}
void MatrixTest(int size, void* func)
{

	int blockSize = 0;
	int gridSize = 0;

	dim3 grid(size, size);

	float *a = new float[size * size];
	float *b = new float[size * size];
	float *c = new float[size * size];


	float *dev_a = nullptr;
	float *dev_b = nullptr;
	float *dev_c = nullptr;

	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&dev_a, size * size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");

	}

	cudaStatus = hipMemcpy(dev_a, a, size * size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");

	}

	cudaStatus = hipMalloc((void**)&dev_b, size * size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");

	}

	cudaStatus = hipMemcpy(dev_b, b, size * size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");

	}

	cudaStatus = hipMalloc((void**)&dev_c, size * size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");

	}

	cudaStatus = hipMemcpy(dev_c, c, size * size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");

	}
	
	getOccupancy(size, gridSize, blockSize, func);
	

	MatrixInicialization <<<1, grid >>> (dev_a);
	MatrixInicializationBlock << <grid, 1 >> > (dev_b);
	MatrixTranspose << <1, grid >> > (dev_c,dev_a);
	hipDeviceSynchronize();

	cudaStatus = hipMemcpy(a,dev_a, size * size  * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");

	}
	cudaStatus = hipMemcpy(b, dev_b, size * size * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");

	}
	cudaStatus = hipMemcpy(c, dev_c, size * size * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");

	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	ViewMatrix(c,size);
}

void ViewMatrix(float *a,int size)
{
	cout << endl;
	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
			if((j + i * size) % size == 0)
				cout << a[j + i * size] << " ";
			else
				cout << "\t" << a[j + i * size] << " ";

		cout << endl;
	}
				
	cout << endl;
}