#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <ctime>
#include <stdio.h>
#include <cstdlib>
#include <iostream>

#include <clocale>

//������ ��� ��������� ������
#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}
//���������

const int countThreads = 1024; //��� �� �����
const int countBlocks = 1;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size,unsigned int countBlock,unsigned int countThread);

static void RunVectorSumm(int sizeVector, int countBlock, int countThread);

//������� ������������ �� GPU
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
	int x;
	setlocale(LC_ALL,"Russian");
	srand(time(NULL));
	//���� �� ����������� ������� �� 10 �� 24
	for (int i = 10; i < 24; i++)		
		RunVectorSumm(i, countBlocks, countThreads);
	
    return 0;
}

void RunVectorSumm(int sizeVector, int countBlock, int countThread)
{
	//������������� �������
	int arraySize = sizeVector;
	int *a  = new int[arraySize];
	int *b = new int[arraySize];
	int *c = new int[arraySize];

	for (int i = 0;i < arraySize;i++)
	{
		a[i] = rand();
		b[i] = rand();
	}
	
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize,countBlock,countThread);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return;
	}
	//printf(" %d" + c[0]);
	
	/*printf("{");
	for (int i = 0; i < arraySize; i++)
	{
		printf(" %d" + c[i]);
	}
	printf("}");
	printf("\n");*/

	//������ ������� ���������� ��������� ��������� ����� ������� ��� ���������� ������ ������������ �������������� � ������������.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return;
	}
}
// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size, unsigned int countBlock, unsigned int countThread)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaCheckError(hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice));
    
	
	//������ �������� Event
	hipEventRecord(start, 0);
    
	//��������� ���������� �� GPU countBlock - ���������� ������(������) , countThread - ���������� �����
    addKernel<<<countBlock, countThread >>>(dev_c, dev_a, dev_b);
	hipEventRecord(stop, 0);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    

	cudaCheckError(hipDeviceSynchronize());
 
	cudaCheckError(cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost));
   
	
	//��������� �������� Event
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	std::cout << std::endl;
	printf("����� ����������: %f ms\n", time);

	//����� ���������� ����������
	/*
	printf("{");
	for (int i = 0; i < size; i++)
	{
		printf(" %d", a[i]);

	}
	printf("} + ");
	printf("{");
	for (int i = 0; i < size; i++)
	{

		printf(" %d", b[i]);

	}
	printf("} = ");
	printf("{");
	for (int k = 0; k < size; k++)
	{
		printf(" %d", c[k]);
	}
	printf("}");*/
	
Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
