#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <clocale>
#include <stdio.h>


int arraySize = 36;

const int countThreads = 1024; //��� �� �����
const int countBlocks = 1;
//������ ��� ��������� ������
#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}

__global__ void gTest1(float* a)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int l = gridDim.x * blockDim.x;

	a[i + j * l] = (float)(threadIdx.x + blockDim.y * blockIdx.x);
}

__global__ void gTest2(float* a)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int l = gridDim.y * blockDim.y;

	a[i + j * l] = (float)(threadIdx.x + blockDim.y * blockIdx.x);
}
//���������

const int countThreads = 1024; //��� �� �����
const int countBlocks = 1;


int warpSize;
int maxThreadsPerMultiProcessor;
int maxThreadsPerBlock;


void getViewGPUSpec();

using namespace std;

int main()
{
	setlocale(LC_ALL,"Russian");
	getViewGPUSpec();

	int *a = new int[arraySize];
	int *dev_a = 0;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		
	}
	cudaStatus = hipMalloc((void**)&dev_a, arraySize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		
	}

	cudaStatus = hipMemcpy(dev_a, a, arraySize * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		
	}

	gTest1 <<<countBlock, countThread >>> (dev_a);







    return 0;
}

void getViewGPUSpec()
{
	int count = 0;
	int dev;
	hipDeviceProp_t prop;

	hipError_t error_id = hipGetDeviceCount(&count);//�������� ���������� ��������� GPU

	if (error_id != hipSuccess) {
		printf("hipGetDeviceCount returned %d\n-> %s\n",
			static_cast<int>(error_id), hipGetErrorString(error_id));
		printf("Result = FAIL\n");
		exit(EXIT_FAILURE);
	}

		

		for (dev = 0; dev < count; ++dev) {

			hipSetDevice(dev);
			hipGetDeviceProperties(&prop, dev);
			cout << "��������� ���������� �� ���������� : " << prop.name << endl;
			cout << "����� ���������� ������ : " << prop.totalGlobalMem / 1024 / 1024 << " MB" << endl;
			cout << "����� ����������� ������ � ����� ����� : " << prop.sharedMemPerBlock << " ���� " << endl;
			cout << "���������� ����� � �����  : " << prop.warpSize << endl;
			cout << "���������� �����������������  : " << prop.multiProcessorCount << endl;
			cout << "����� ����������� ������  : " << prop.totalConstMem << endl;
			cout << "������������ ���������� ����������������� �� ���� : " << prop.maxThreadsPerMultiProcessor << endl;
			cout << "������������ ���������� ������� �� ����  : " << prop.maxThreadsPerBlock << endl;

		}
}

